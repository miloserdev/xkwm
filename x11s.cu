//#include <bits/signum-arch.h>
//#include <bits/signum-generic.h>
//#include <bits/types/FILE.h>
#define N 10000000

#define __GNU_SOURCE

#include <hip/hip_runtime.h>
#include <dlfcn.h>

#include <stdarg.h>
#include <signal.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <sys/wait.h>
#include <unistd.h>
#include <X11/X.h>
#include <X11/cursorfont.h>
#include <X11/keysymdef.h>
#include <X11/Xatom.h>
#include <X11/Xlib.h>
#include <X11/Xutil.h>

#ifdef XINERAMA
#include <X11/extensions/Xinerama.h>
#endif

#include <sys/types.h>
#include <fcntl.h>

#include <setjmp.h>
#define try int __error=-1; 		//bool __HadError=false;
#define catch(x) ExitJmp:if(__error > -1)
#define throw(x) {__error = x; goto ExitJmp;}


void kz(int x) {
	if ( signal(SIGCHLD, kz) == SIG_ERR ) { exit(-1); }
	while (0 < waitpid(-1, NULL, WNOHANG) );
}


FILE *fp;

void _log( const char *fmt, ... ) {
	fp = fopen("fuck.log", "a");
	if (fp == NULL) { exit(1); }
	va_list args;
	va_start(args, fmt);
	vfprintf(fp, fmt, args);
	va_end(args);
	fclose(fp);
}


int lerp(int a, int b, int t) {
	return a + t * (b - a);
}





#include <linux/cuda.h>
__global__ void interpolate(int count, float *result) {

	int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (globalIdx < count) {
		result[globalIdx] = result[globalIdx] * 9.5f;
	}

//	int x = 1;
//	int i = 0;

/*	try {
	//	throw(0);
	//	*((unsigned int*)0) = 0xDEAD;
		_log("__S1_%d__", 1);
	} catch (x) {
		_log("Error occured %d", __error);
	}
*/

//	int x = 18874374;

//	char fpp = dlsym(RTLD_NOW, "XCreateWindow");
	//Window fptr = funcptr(0, 2354);

/*	for (;;) {
		_log("__________%s______________I_%d___WINDOW_%d___FPTR_%c__________\n", __FUNCTION__, i, x, 'a');
//		x = x + 2097152;
		x = x + 2097152;
		i++;
		if (x > 999999999) return;
	}
*/

//	free(i);
//	free(to);
//	free(last);
}

static int (*xerrorxlib)(Display*, XErrorEvent *);

static int xerrorstart(Display *display, XErrorEvent *e) {
	_log("another instance" -1);
	return -1;
}

static int xerror(Display *display, XErrorEvent *e) {
	_log("request code %d, error code: %d\n", e->request_code, e->error_code);
	return xerrorxlib(display, e);
}

static int xerrordummy(Display *display, XErrorEvent *e) {
	_log("xid: %s, request code: %d, error code: %d...idle....   \n", e->resourceid, e->request_code, e->error_code);
}

unsigned long _RGB(int r, int g, int b) {
	return b + (g << 8) + (r << 16);
}


//static Display *display;
Display *display;
static Window root;
int rootw;

static int def_screen;
static int sw, sh;

unsigned long black, white;

static Window checkwin;

Font font;
Cursor cursor, cursor_move, cursor_size;

int border_width = 1;
int border_color;
int border_color_focused;
bool fs_border = true;

int mon_count = 3;
int curr_mon = 1;


typedef struct program {
	Window window;
	int id;
	int monitor;
	bool raised;
	bool fullscreen;

	GC gc;

	int x, y;
	int width, height;
	int x_fs, y_fs, wid_fs, hei_fs;
	int x_rs, y_rs, wid_rs, hei_rs;

	struct program *next;
	struct program *prev;
} program;


struct program *last, *first;
int prog_count = 0;


void init_prog() {
	last = (struct program*) malloc(sizeof(struct program));
	last->id = 0;
	last->window = root;
	last->monitor = 0;
	last->next = last;
	last->prev = last;
	first = last;
	_log("__________%s______________ROOT_WINDOW_INIT_%d___MONITOR_%d__________\n", __FUNCTION__, last->window, last->monitor);

	prog_count++;
}

program * get_prog(Window win) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, win);

	program *prog = first;
	for (;;) { //for (int p = 0; p <= prog_count; p++) {
		prog = prog->next;
		if (prog->window == win) return prog;
		if (prog == first) break;
	}

	if (win == 1) return first;

	_log("__________%s______________WINDOW_%d___DOES_NOT_EXIST__________\n", __FUNCTION__, win);

	return NULL;
}

program * add_prog(Window win) {

	_log("__________%s____________________RTLD_%d__________\n", __FUNCTION__, RTLD_NOW);

	if (win == root || win == rootw || win == 1) {
		_log("__________%s______________IMPOSTOR______WINDOW_%d__________\n", __FUNCTION__, win);
		return NULL;
	}

	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, win);

	XWindowAttributes xwa;
	if (!XGetWindowAttributes(display, win, &xwa)) return NULL;
	//XGetWindowAttributes(display, win, &xwa);

	struct program *obj;
	obj = (struct program*) malloc(sizeof(struct program));

	bool exist = false;

	program *prog = first;
	for (;;) {
		prog = prog->next;
		if (prog->window == win) exist = true;
		if (prog == first) break;
	}

	//if ( (get_prog(win) == NULL) ) {
	if ( !exist ) {
		_log("__________%s_________BEGIN_____WINDOW_%d__________\n", __FUNCTION__, win);
		obj->id = prog_count;
		obj->window = win;
		obj->monitor = curr_mon;
		obj->raised = true;
		obj->fullscreen = false;

		obj->x = xwa.x;
		obj->y = xwa.y;
		obj->width = xwa.width;
		obj->height = xwa.height;
		obj->x_rs = 0;
		obj->y_rs = 0;
		obj->x_fs = 0;
		obj->y_fs = 0;

		obj->next = first;
		obj->prev = last;

		last->next = obj;
		last = last->next;


		prog_count++;

		_log("__________%s______________WINDOW_%d___PROG_ADDED_%d__________\n", __FUNCTION__, win, obj->id);
		return obj;

	} else {
		_log("__________%s______________WINDOW_%d_EXIST_________\n", __FUNCTION__, win);
	}
	return NULL;
}


void rem_prog(Window win) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, win);

	program *prog = first;

	for (;;) { //for (int i = 0; i <= prog_count; i++) {
		prog = prog->next;
		_log("__________%s______________REMOVE_ITERATION___WINDOW_%d__________\n", __FUNCTION__, prog->window);

		if (prog->window == win) {
			_log("__________%s______________WINDOW_%d___REMOVING___NEXT_%d___PREV_%d___LAST_%d_____________\n", __FUNCTION__, win, prog->next->window, prog->prev->window, last->window);

			prog->next->prev = prog->prev;
			prog->prev->next = prog->next;
			last = prog->next != first ? prog->next : prog->prev;

			_log("__________%s______________WINDOW_%d___REMOVING___NEXT_%d___PREV_%d___LAST_%d_____________\n", __FUNCTION__, win, prog->next->window, prog->prev->window, last->window);

			_log("__________%s______________WINDOW_%d___REMOVED___ID_%d__________\n", __FUNCTION__, win, prog->id);
			break;

//			if (prog_count > 1) prog_count--;
		}
		if (prog == first) break;

	}

/*	if ( (prog = get_prog(win)) ) {
		prog->next->prev = prog->prev;
		prog->prev->next = prog->next;
		last = prog->next;
		prog_count--;
		_log("__________%s______________WINDOW_%d___PROG_REMOVED_%d__________\n", __FUNCTION__, win, prog->id);
	}
*/
//	for (int i = 0; i <= prog_count; i++) {
//		_log("__________%s______________ITERATION_PROG_%d__________\n", __FUNCTION__, prog->window);
//	}
}


void update_border(Window win, int width, int color) {
	XSetWindowBorderWidth(display, win, width);
	XSetWindowBorder(display, win, color);
}


void update_bordered(Window win) {
	update_border(win, border_width, border_color);
}


Window get_focused() {
	_log("__________%s______________________\n", __FUNCTION__);
	Window winfoc;
	int rev_foc;
	XGetInputFocus(display, &winfoc, &rev_foc);
	_log("__________%s_____________WINDOW_FOCUSED_%d__________\n", __FUNCTION__, winfoc);
	return winfoc;
}


void update_focused() {
	Window focused = get_focused();
	if (focused == root || focused == 1) return;

	program *upd = first;
	for (;;) {
		upd = upd->next;
		if (upd->monitor == curr_mon) {
			if (upd->window == focused) {
				update_border(upd->window, border_width, border_color_focused);
			} else {
				update_border(upd->window, border_width, border_color);
			}
		}
		if (upd == first) break;
	}
}


void focus(Window window) {
	_log("__________%s_____________WINDOW_%d__________\n", __FUNCTION__, window);

	update_focused();

	XRaiseWindow(display, window);
	XSetInputFocus(display, window, RevertToPointerRoot, CurrentTime);

	update_border(window, border_width, border_color_focused );
}


void change_program(Window win, int mon) {
	_log("__________%s______________MONITOR_%d__________\n", __FUNCTION__, mon);

	program *prog = get_prog(win);
	update_bordered(prog->window);

	int count = 0;
	for (;;) {//for (int i = 0; i <= prog_count; i++) {
		prog = prog->next;
		_log("__________%s______________PROG_%d___WINDOW_%d__________\n", __FUNCTION__, prog->id, prog->window);

		if (prog->monitor == mon) {
			if (prog->window != root || prog->window != 1) {
				focus(prog->window);
				break;
			}
		}
	}
}


void counter() {
	Window focused = get_focused();
	int count = 0;
	program *prog = first;
	for (;;) {
		prog = prog->next;
	//	_log("__________%s______________COUNT___WINDOW_%d___ID_%d__________\n", __FUNCTION__, prog->window, prog->id);

		_log("			__________%s______________COUNT___WINDOW_%d___ID_%d___NEXT_%d___PREV_%d___LAST_%d_____________\n", __FUNCTION__, prog->window, prog->id, prog->next->window, prog->prev->window, last->window);

		count++;
		if (prog == first) break;

	}
	_log("			__________%s______________COUNT_%d__________\n", __FUNCTION__, count);


	_log("			__________%s______________FOCUSED_%d__________\n", __FUNCTION__, focused);

}


void stack_programs(int mon) {
	_log("__________%s_____________MONITOR_%d__________\n", __FUNCTION__, mon);

	int gap = 10;
	int _sw = sw, _sh = sh;
	int _bw = border_width + gap;
	int _x = 0 - _bw + gap, _y = 0 - _bw + gap;
	int ci = 1; //2;

	int count = 0;
	int progs = 0;
	program *prog = first; // last;
	for (;;) { //for (int i = 0; i <= prog_count + 10; i++) {
		prog = prog->next;
		progs++;
		if (prog->monitor == mon) {
			_log("__________%s_________________CURRENT_%d___ROOT_%d__________\n", __FUNCTION__, prog->window, root);
			count++;
		}
		if (prog == first) break;

	}

	_log("__________%s_________________PROGS_ALL_%d___PROGS_AT_SCREEN_%d__________\n", __FUNCTION__, progs, count);


	prog = first;

	for (;;) {
		prog = prog->next;

		_log("__________%s_________________NEXT_%d__________\n", __FUNCTION__, prog->window);

		if (prog->window != root && prog->monitor == curr_mon) {

			if (ci % 2 != 0) { _sw = _sw / 2; if (ci >= count) _sw = _sw * 2;
			} else { _sh = _sh / 2; if (ci >= count) _sh = _sh * 2; }

			XMoveResizeWindow(display, prog->window, _x + _bw, _y + _bw, _sw - _bw * 2, _sh - _bw * 2);
			_log("__________%s_________________WINDOW_%d_STACKED_X_%d_Y_%d_WIDTH_%d_HEIGHT_%d_________\n", __FUNCTION__, prog->window, _x + _bw, _y + _bw, _sw - _bw * 2, _sh - _bw * 2);

			if (ci % 2 != 0) { _x = _x + _sw;
			} else { _y = _y + _sh; }
			ci++;
		}
		if (prog == first) break;
	}
}


void switch_program(Window win) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, win);

	counter();

	program *prog1 = get_prog(win);
	program *prog2 = first->next;

	_log("__________%s______________SWITCH_%d_ID_%d___TO_%d_ID_%d__________\n", __FUNCTION__, prog1->window, prog2->window);

	int temp1_id = prog1->id;
	int temp1_window = prog1->window;

	int temp2_id = prog2->id;
	int temp2_window = prog2->window;

	prog2->id = prog1->id;
	prog2->window = prog1->window;

	prog1->id = temp2_id;
	prog1->window = temp2_window;


	_log("__________%s______________SWITCHED_%d_ID_%d___TO_%d_ID_%d__________\n", __FUNCTION__, prog1->window, prog2->window);

	counter();

//	change_program(win, prog->monitor);
	focus(temp1_window);
	stack_programs(prog1->monitor);
}


void fullscreen(Window win) {

	if (win == root || win == 1) return;

	program *prog = get_prog(win);

	if ( prog->window != root && win != 1) {
		XWindowAttributes xwa;
		XGetWindowAttributes(display, prog->window, &xwa);

		if (prog->monitor == curr_mon) {
			if (!prog->fullscreen) {

				if (!prog->fullscreen) {
					prog->x_fs = xwa.x;
					prog->y_fs = xwa.y;
					prog->wid_fs = xwa.width;
					prog->hei_fs = xwa.height;
				}

				int wid = fs_border ? sw - (border_width * 2) : sw;
				int hei = fs_border ? sh - (border_width * 2) : sh;

				int wwid = 100;
				int whei = 100;
				int wx = 500;
				int wy = 500;

			//	XSetWindowBorderWidth(display, prog->window, 
			//		(fs_border ? border_width : 0)
			//	);
				update_border(prog->window, (fs_border ? border_width : 0), border_color);
				XMoveResizeWindow(display, prog->window, 0, 0, wid, hei);
				prog->fullscreen = true;
			} else {
				update_bordered(prog->window);
			//	XSetWindowBorderWidth(display, prog->window, border_width);
				XMoveResizeWindow(display, prog->window, prog->x_fs, prog->y_fs, prog->wid_fs, prog->hei_fs);
				prog->fullscreen = false;
			}
		}

		focus(prog->window);
	}
}


void move_resize_window(XButtonEvent *e) {
	_log("__________%s_____________CALLER_%d__________\n", __FUNCTION__, e->window);

	Window win = get_focused();
	_log("__________%s_____________FOCUSED_%d__________\n", __FUNCTION__, win);

	XWindowAttributes xwa;
	XGetWindowAttributes(display, win, &xwa);

	int wid = xwa.width, hei = xwa.height;
	int rx, ry, cx, cy = 0;
	unsigned int mask = 0;
	Window c, r;

	XQueryPointer(display, win, &r, &c, &rx, &ry, &cx, &cy, &mask);

	XGrabPointer(display, root, True, ButtonPressMask | ButtonReleaseMask |
		ButtonMotionMask | PointerMotionMask, GrabModeAsync, GrabModeAsync, None,
		(e->button == Button1 ? cursor_move : cursor_size), CurrentTime);
	XEvent ev;

	do {
		XMaskEvent(display, ButtonPressMask | ButtonReleaseMask | ButtonMotionMask | PointerMotionMask | ExposureMask | SubstructureRedirectMask, &ev);
		switch (ev.type) {
			case Expose:
			case MapRequest:
			case ConfigureRequest:
				break;
			case MotionNotify:
				XSetTransientForHint(display, root, win);
				if (e->button == Button1) {
					XMoveWindow(display, win, xwa.x + (ev.xmotion.x - rx), xwa.y + (ev.xmotion.y - ry));
				} else {
					wid = ev.xmotion.x - xwa.x;
					hei = ev.xmotion.y - xwa.y;
					XResizeWindow(display, win, wid, hei);
				}
				break;
		}
	} while (ev.type != ButtonRelease);
	XUngrabPointer(display, CurrentTime);
}


void handle_expose(XExposeEvent *e) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, e->window);
	update_focused();
	XSync(display, False);
}


void handle_map_request(XMapRequestEvent *e) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, e->window);

	XWindowAttributes xwa;
	if (!XGetWindowAttributes(display, e->window, &xwa) || xwa.override_redirect) return;

	XSetWindowAttributes at;
	at.event_mask = SubstructureRedirectMask |
			SubstructureNotifyMask |
			StructureNotifyMask |
			ExposureMask |
			KeyPressMask |
			KeyReleaseMask |
			ButtonPressMask |
			ButtonReleaseMask |
			EnterWindowMask |
			LeaveWindowMask |
			FocusChangeMask ;

//	XChangeWindowAttributes(display, e->window, CWEventMask | CWCursor, &at);
	XSelectInput(display, e->window, EnterWindowMask | FocusChangeMask |
		PropertyChangeMask | StructureNotifyMask);

//	XSetWindowBorderWidth(display, e->window, border_width);
//	XSetWindowBorder(display, e->window, border_color);
	update_bordered(e->window);

	program *prog = add_prog(e->window);

	XGCValues xgc;
	xgc.foreground = white;
	xgc.background = black;

	GC gc = XCreateGC(display, prog->window, GCForeground | GCBackground, &xgc);
	prog->gc = gc;
	XFillRectangle(display, prog->window, gc, 100, 100, 200, 200);


	XMapWindow(display, e->window);
	focus(e->window);
}


//key_release
void handle_key_press(XKeyEvent *e) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, e->window);
}


//key_press
void handle_key_release(XKeyEvent *e) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, e->window);

	Window focused = get_focused();

	counter();

	int _mask = (e->state & (ShiftMask | ControlMask | Mod1Mask | Mod3Mask | Mod4Mask | Mod5Mask) );

	_log("__________%s______________MASK_%d___MASKCTRL_%d__________\n", __FUNCTION__, _mask, ControlMask);

	switch (XKeycodeToKeysym(display, e->keycode, 0)) {

		case XK_Tab:
			if (_mask == (Mod1Mask | ControlMask) ) {
				switch_program(focused);
			} else {
				change_program(focused, curr_mon);
			}
			break;
		case XK_Return:
			fullscreen(focused);
			break;
		case XK_5:
			stack_programs(curr_mon);
			break;
		case XK_7:
			XLowerWindow(display, focused);
			break;
		case XK_0:
			if (e->state & Mod1Mask) {
				if (fork() == 0) {
					if (display) close(ConnectionNumber(display));
					char *cmd[] = { "dmenu_run", NULL };
					setsid();
					execvp(cmd[0], cmd);
					perror("failed");
					exit(EXIT_SUCCESS);
				}
			}
			break;
	}
}


void handle_configure_request(XConfigureRequestEvent *e) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, e->window);

	Window focused = get_focused();

	XWindowAttributes xwa;
	XGetWindowAttributes(display, e->window, &xwa);

	long msize;
	XSizeHints size;
	if (!XGetWMNormalHints(display, e->window, &size, &msize)) {
		size.flags = PSize;
	}

	XWindowChanges wcs;
	wcs.x = e->x;
	wcs.y = e->y;
	wcs.width = e->width;
	wcs.height = e->height;
	wcs.border_width = border_width;
	wcs.sibling = e->above;
	wcs.stack_mode = e->detail;
	XConfigureWindow(display, e->window, e->value_mask, &wcs);

//	update_border(e->window);
	XSync(display, False);
}


void handle_configure_notify(XConfigureEvent *e) {
	program *prog = get_prog(e->window);
	XFillRectangle(display, prog->window, prog->gc, 100, 100, 200, 200);

	update_focused();
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, e->window);
}


void handle_button_press(XButtonEvent *e) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, e->window);

	// if (e->window == root) return;

	move_resize_window(e);
}


void handle_button_release(XButtonEvent *e) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, e->window);
}


void handle_unmap_notify(XUnmapEvent *e) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, e->window);
//	rem_prog(e->window);
	XUnmapWindow(display, e->window);
}


void handle_destroy_notify(XDestroyWindowEvent *e) {
	_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, e->window);
	rem_prog(e->window);
}

void gpuscc() {
	_log("GPU -> init all\n");

	int count = 1000;
	float *result, *d_result;
	result = (float *) malloc(count * sizeof(float));
//	int *result = new int[count];
	for (int i = 0; i < count; i++) {
		result[i] = 1.3f;
	}

	_log("GPU -> result init\n");


	hipMalloc(&d_result, count * sizeof(float));
	hipMemcpy(d_result, result, count * sizeof(float), hipMemcpyHostToDevice);

	_log("GPU -> cuda malloc\n");

	interpolate<<<1,1>>>(count, d_result);

	_log("GPU -> exec void on device\n");

	hipDeviceSynchronize();
	_log("GPU -> syncing\n");

	hipMemcpy(result, d_result, count * sizeof(float), hipMemcpyDeviceToHost);
	_log("GPU -> memcpy from device to host\n");

	hipFree(d_result);
	_log("GPU -> free cuda memory\n");


	_log("GPU complete\n");
	_log("from GPU %d\n", result);
}

int main(void) {

	fp = fopen("fuck.log", "w");
	fprintf(fp, "");
	fclose(fp);

	_log("_____________________________________\n");
	_log("________________1BEGIN________________\n");
	_log("_____________________________________\n");

	gpuscc();


	kz(0);

//	interpolate();

	display = (Display *) XOpenDisplay(NULL);
	def_screen = DefaultScreen(display);

	sw = DisplayWidth(display, def_screen);
	sh = DisplayHeight(display, def_screen);
	root = RootWindow(display, def_screen);
	rootw = XDefaultRootWindow(display);

	black = BlackPixel(display, def_screen);
	white = WhitePixel(display, def_screen);
	border_color = _RGB(100,100,100);
	border_color_focused = _RGB(255,255,255);

	XSync(display, False);
	XFlush(display);

	xerrorxlib = XSetErrorHandler(xerrorstart);
	XSetErrorHandler(xerrordummy);
	XSetIOErrorHandler( (XIOErrorHandler) xerrordummy);



	XSync(display, False);
	//init progs
	init_prog();


	font = XLoadFont(display, "-*-*-*-R-Normal--*-180-100-100-*-*");
	cursor = XCreateFontCursor(display, XC_left_ptr);
	cursor_move = XCreateFontCursor(display, XC_fleur);
	cursor_size = XCreateFontCursor(display, XC_sizing);


//	XSetFont(display, gc, font);
//	Drawable dr;

	char *_num;
	_num = "text";//prog->id;

//	XSetBackground(display, gc, white);
//	XSetForeground(display, gc, black);

//	XDrawString(display, dr, gc, 120, 120, _num, strlen(_num));





	XSetWindowAttributes at;
	at.event_mask = SubstructureRedirectMask |
			SubstructureNotifyMask |
			StructureNotifyMask |
			ExposureMask |
			KeyPressMask |
			KeyReleaseMask |

			PointerMotionMask |
			ButtonPressMask |
			ButtonReleaseMask |

			PropertyChangeMask ;

	at.do_not_propagate_mask = 0;
	at.cursor = cursor;

	unsigned long at_mask = CWEventMask | CWCursor | NoEventMask; // | CWBackPixel; - Causes color blink at start;
	XSelectInput(display, root, at.event_mask);
	XSync(display, False);
	XChangeWindowAttributes(display, root, at_mask, &at);


	XUngrabKey(display, AnyKey, AnyModifier, root);

	XGrabKey(display, XKeysymToKeycode(display, XK_0), Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);
	XGrabKey(display, XKeysymToKeycode(display, XK_5), Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);
	XGrabKey(display, XKeysymToKeycode(display, XK_7), Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);
	XGrabKey(display, XKeysymToKeycode(display, XK_Tab), Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);
	XGrabKey(display, XKeysymToKeycode(display, XK_Return), Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);
	XGrabKey(display, XKeysymToKeycode(display, XK_Control_L), Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);

	XGrabKey(display, XKeysymToKeycode(display, XK_1), Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);
	XGrabKey(display, XKeysymToKeycode(display, XK_2), Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);
	XGrabKey(display, XKeysymToKeycode(display, XK_3), Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);

	XGrabKey(display, XKeysymToKeycode(display, XK_1), ControlMask | Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);
	XGrabKey(display, XKeysymToKeycode(display, XK_2), ControlMask | Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);
	XGrabKey(display, XKeysymToKeycode(display, XK_3), ControlMask | Mod1Mask, root, True, GrabModeAsync, GrabModeAsync);



	XGrabButton(display, Button1, Mod1Mask, root, False, ButtonPressMask | ButtonReleaseMask | ButtonMotionMask, GrabModeAsync, GrabModeAsync, None, None);
	XGrabButton(display, Button3, Mod1Mask, root, False, ButtonPressMask | ButtonReleaseMask | ButtonMotionMask, GrabModeAsync, GrabModeAsync, None, None);



	XSync(display, False);
	XFlush(display);



	XEvent e;
	XSync(display, False);

//asda

	while (1) {

		program *prog = first;
		for (;;) { //for (int p = 0; p <= prog_count; p++) {
			prog = prog->next;

			if (prog != NULL && prog->window != root) XFillRectangle(display, prog->window, prog->gc, 100, 100, 200, 200);

			if (prog == first) break;
		}


		XNextEvent(display, &e);

//		Window ww = get_focused();

		switch (e.type) {
		/*	case FocusIn:
			case FocusOut:
			case GraphicsExpose:
			case NoExpose:
			case VisibilityNotify:
			case UnmapNotify:
			case MapNotify:
			case ReparentNotify:
			case GravityNotify:*/
	/*		case CirculateRequest:
			case CirculateNotify: _log("Circulate\n"); break;
			case PropertyNotify:
				_log("PropertyNotify\n");

				XPropertyEvent *ev;
				ev = &e.xproperty;
				_log("__________%s______________WINDOW_%d__________\n", __FUNCTION__, ev->window);

				program *prog = get_prog(get_focused());//get_prog(ev->window);
				if (prog != NULL && prog->window != root) XFillRectangle(display, prog->window, prog->gc, 100, 100, 200, 200);

				break;
			case MotionNotify: _log("MotionNotify\n"); break;
			case CreateNotify: _log("CreateNotify\n"); break;
			case VisibilityNotify: _log("VisibilityNotify\n"); break;
			case GraphicsExpose: _log("GraphicsExpose\n"); break;
			case NoExpose: _log("NoExpose\n"); break;
			case ClientMessage: _log("ClientMessage\n"); break;

			case ColormapNotify: _log("ColormapNotify\n"); break;
			case SelectionNotify:
			case SelectionRequest:
			case SelectionClear:
				 _log("SelectionNotify\n"); break;
			case MappingNotify: _log("MappingNotify\n"); break;
			case GravityNotify: _log("GravityNotify\n"); break;
			case ReparentNotify: { _log("ReparentNotify\n"); } break;

		//	case AnyEvent:
		//		program *prog = get_prog(&e.xany->window);
		//		if (prog != NULL) XFillRectangle(display, prog->window, prog->gc, 100, 100, 200, 200);
		//		break;
	*/
			case Expose: {
				handle_expose(&e.xexpose);
				}
				break;
			case KeyPress: {
				handle_key_press(&e.xkey);
				}
				break;
			case KeyRelease: {
				handle_key_release(&e.xkey);
				}
				break;
			case ConfigureRequest:
				handle_configure_request(&e.xconfigurerequest);
				break;
			case ConfigureNotify:
				handle_configure_notify(&e.xconfigure);
				break;
			case MapRequest:
				handle_map_request(&e.xmaprequest);
				break;
			case ButtonPress:
				handle_button_press(&e.xbutton);
				break;
			case ButtonRelease:
				handle_button_release(&e.xbutton);
				break;
			case DestroyNotify:
				handle_destroy_notify(&e.xdestroywindow);
				break;
			default:
				_log("__________%s______________UNKNOWN_EVENT_%d__________\n", __FUNCTION__, e.type);
				break;
		}
	}
	return 0;
}












/*
jmp_buf ret[16];
int rs = -1;
#define endtry rs--
#define try(e) rs++; if ((e = setjmp(ret[rs])) == 0)
#define catch else
#define perror(e) printf("Netsing level %d: error %dn", rs, e);
*/

/*
#define try do { jmp_buf ex_buf__; switch( setjmp(ex_buf__) ) { case 0: while (1) {
#define catch(x) break; case x:
#define finally break; } default: {
#define etry break; } } } while(0)
#define throw(x) longjmp(ex_buf__, x)
*/



/*
jmp_buf *g__ActiveBuf;
#define try jmp_buf __LocalJmpBuf; jmp_buf *__OldActiveBuf=g__ActiveBuf;bool __WasThrown=false;g__ActiveBuf=&__LocalJmpBuf;if(setjmp(__LocalJmpBuf)){__WasThrown=true;}else
#define catch(x) g__ActiveBuf=__OldActiveBuf;if(__WasThrown)
#define throw(x) longjmp(*g__ActiveBuf, 1);
*/
